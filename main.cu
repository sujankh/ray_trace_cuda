#include "hip/hip_runtime.h"
#include <stdio.h>
#include <algorithm>
#include <iostream>
#include <cmath>

#include <thrust/device_ptr.h>
#include <thrust/extrema.h>
#include <thrust/functional.h>
#include <thrust/transform_reduce.h>


#include "vec3.h"

// Mathematics obtained from Fundamentals of Computer Graphics (3rd edition) book

struct sphere;
struct scene;

struct vec2
{
    int x, y;
};

struct material
{
    vec3 color;
    // TODO: properties like phong exponent, specular coeffecient etc
};

struct light_source
{
    vec3 position;
    vec3 intensity;
};

struct image_plane
{
    int l, r; // Left, right edges of the image plane in 3D world
    int t, b; // Top, bottom edges of the image plane
    int nx, ny; // The dimension in pixels of the plane
    float distance; //Distance from the camera to the image_plane
};

struct hit_info
{
    const sphere *obj;
    float t;
    vec3 hit_point;  // Point in surface of obj where the ray hits
};


//
// Represents a ray p(t) = e + t * d
// where e = origin of the ray
// and d = direction of the ray
//
struct ray
{
    vec3 d; // direction
    vec3 e; // origin of the ray

    //
    // Get the point in the ray at parametric value t
    //
    __host__ __device__
    vec3 get_point(float t) const
    {
        return e + t * d;
    }
};

struct sphere
{
    sphere(vec3 c, float radius): center(c), R(radius)
    {
    }

    vec3 center;
    float R; // radius
    material m;

    //
    // Returns if the ray hits an object in the scene
    // Updates at what value of t, the ray hits this object
    //
    __device__ bool hits(const ray& r, hit_info* hit) const
    {
        // Value of t for a parametric representation of the ray p(t) = e + td
        // where vectors e = camera, d = ray

        // Intersection of ray with sphere: c = center of sphere
        // t` = -d . (e-c) +- sqrt((d.(e-c))^2 - (d.d) ((e-c).(e-c) - R*2))
        // t = t`/(d.d)

        vec3 ce = r.e - center;
        float d_d = dot(r.d, r.d);
        float d_ce = dot(r.d, ce);

        float discriminant = d_ce * d_ce - d_d * (dot(ce, ce) - R * R);
        if(discriminant >= 0)
        {
            // Update hit_info
            hit->obj = this;

            float discriminant_sqrt = std::sqrt(discriminant);
            // The tracer only cares about the point at which the ray enters the sphere
            // So just set t as the smallest t obtained
            hit->t = fminf((-1 * d_ce + discriminant_sqrt)/d_d, (-1 * d_ce - discriminant_sqrt)/d_d);
            return true;
        }
        else
        {
            return false;
        }
    }

    //
    // Compute the normal on the surface at the given point
    //
    __device__ vec3 normal(vec3 point) const
    {
        // Expecting point to always be on the surface of the sphere
        return (point - center) / R;
    }

};


struct scene
{
    // The world is a list of spheres for now
    // There are some issues in CUDA with polymorphic types
    // https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#virtual-functions
    // Refer to branch renderable_abstract_class for more details
    sphere* world;
    int num_objects;
    vec3 background;

    image_plane image;
    vec3 camera;
    light_source light;
    light_source ambient;

    //
    // Returns if the ray hits an object in the scene
    // Updates the information about the object in nearest_hit if provided
    //
    __device__
    bool hit(const ray &r, hit_info *nearest_hit)
    {
        float t = 1000000000;
        bool any_hit = false;

        // Check if the ray from the pixel hits any objects in the world
        for (int k = 0; k < num_objects; k++)
        {
            const sphere &obj= world[k];
            hit_info hit;

            // Get the nearest object which the ray p(t) = e + td hits
            // The smaller the value of t, the nearest the object is to the image_plane
            if(obj.hits(r, &hit))
            {
                // If the ray hits the object from which it originates(t = 0) or behind the object (t < 0),
                // we do not want to consider such cases
                if(hit.t <= 0)
                {
                    continue;
                }

                any_hit = true;
                if(nearest_hit == nullptr)
                {
                    // User just want to know if any object is hit
                    break;
                }

                if(hit.t < t)
                {
                    t = hit.t;
                    *nearest_hit = hit;
                }
            }
        }

        return any_hit;
    }
};


//
// Compute the ray passing through the camera and the pixel's center
// Pixel's position specified by (i, j)
//
__device__
ray ray_at_pixel(int i, int j, const image_plane &image, const vec3& camera)
{
    float u = image.l + (image.r - image.l) * (i + 0.5)/image.nx;
    float v = image.b + (image.t - image.b) * (j + 0.5)/image.ny;

    // Ray from camera towards the pixel (negative w for the direction)
    // Right handed co-ordinate system u,v,w
    ray camera_ray;
    camera_ray.d = unit_vector(vec3(u, v, -1 * image.distance));
    camera_ray.e = camera; // The ray originated from the camera
    return camera_ray;
}


//
// Compute the color of the pixel at the point where the ray hits the object
//
__device__ vec3 surface_color(scene *scene, hit_info *hit)
{
    vec3 normal = hit->obj->normal(hit->hit_point);
    ray light_ray;
    light_ray.e = hit->hit_point;
    light_ray.d = unit_vector(scene->light.position - hit->hit_point); // vec(AB) = B - A

    // Use ambient + lambertian shading model: L = ka * Ia + kd * I * max(0, n.l)
    vec3 color = scene->ambient.intensity * hit->obj->m.color;

    if(scene->hit(light_ray, nullptr) == false)  // light ray should not hit any object in the scene for full color
    {
        color += hit->obj->m.color * scene->light.intensity * fmaxf(0, dot(normal, light_ray.d));
    }

    return color;
}

//
// Main kernel: traces a ray into the scene for a pixel specified by thread ID
//
__global__ void trace_ray(vec2* data, vec3* frame_buffer, scene *sc)
{
    int thread_row = blockIdx.y * blockDim.y + threadIdx.y;
    int thread_col = blockIdx.x * blockDim.x + threadIdx.x;

    int index = thread_row * (gridDim.x * blockDim.x) + thread_col;

    int nx = sc->image.nx;
    int ny = sc->image.ny;

    if (index < nx * ny)
    {
        int i  = data[index].x;
        int j = data[index].y;

        const ray& camera_ray = ray_at_pixel(i, j, sc->image, sc->camera);
        vec3 pix_color = sc->background;

        hit_info nearest_hit;

        if(sc->hit(camera_ray, &nearest_hit))
        {
            nearest_hit.hit_point = camera_ray.get_point(nearest_hit.t);
            pix_color = surface_color(sc, &nearest_hit);
        }
        frame_buffer[index] = pix_color;
    }
    else
    {
        printf("Index %d is out of bounds", index);
    }
}


// Unary reduction op to get the max value of either r/g/b for a vec3
struct max_color
{
    __host__ __device__
    float operator()(const vec3& v)
    {
        return thrust::max(thrust::max(v.r(), v.g()), v.b());
    }

};


// Normalizer to normalize samples down to [0, 1] and then scale
struct normalize_color
{
    const float normalizer, scale;

    normalize_color(float _normalizer, int _scale): normalizer(_normalizer), scale(_scale) {}

    int operator()(const float &val) const
    {
        return static_cast<int>(val/normalizer * scale);
    }
};

int main(void)
{
    int nx = 1024;
    int ny = 1024;

    int num_pixels = nx * ny;
    dim3 threads_per_block(16, 16);
    dim3 num_blocks(nx/threads_per_block.x, ny/threads_per_block.y);

    vec3* frame_buffer;
    vec2* data;
    hipMallocManaged(&frame_buffer, num_pixels * sizeof(vec3));
    hipMallocManaged(&data, num_pixels * sizeof(data));

    for(int j = ny -1, k = 0; j >=0; j--, k++)
    {
        for(int i = 0; i < nx; i++)
        {
            int index = i + k * nx;
            data[index].x = i;
            data[index].y = j;
        }
    }

    int num_objects = 4;
    sphere* spheres;
    hipMallocManaged(&spheres, num_objects * sizeof(sphere));

    spheres[0] = sphere(vec3(-4, 1, -10), 1.2);
    spheres[0].m.color = vec3(0.9, 0, 0);

    spheres[1] = sphere(vec3(3.5, 0, -10), 1.4);
    spheres[1].m.color = vec3(0.8, 0.8, 0.8);

    // This sphere is slightly hidden behind the 2nd sphere
    spheres[2] = sphere(vec3(2, 0, -12), 1.5);
    spheres[2].m.color = vec3(0, 0.6, 0);

    // Render a base surface as a big sphere
    // TODO: draw a plane instead
    spheres[3] = sphere(vec3(0, -84, -50), 90);
    spheres[3].m.color = vec3(0.7, 0.7, 0.7);

    image_plane image;
    image.l = -4; image.r = 4;
    image.t = 4; image.b = -4;
    image.nx = nx; image.ny = ny;
    image.distance = 4;

    scene *sc;
    hipMallocManaged(&sc, sizeof(scene));
    sc->background = vec3(0.1, 0.1, 0.1);
    sc->world = spheres;
    sc->num_objects = num_objects;
    sc->camera = vec3(0, 0, 0);
    sc->image = image;

    sc->light.position = vec3(-20, 10, -4);
    sc->light.intensity = vec3(1, 1, 1);
    sc->ambient.intensity = vec3(0.3, 0.3, 0.3);

    trace_ray<<<num_blocks, threads_per_block>>>(data, frame_buffer, sc);
    hipDeviceSynchronize();

    hipError_t error = hipGetLastError();
    if(error!=hipSuccess)
    {
        fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
        return 0;
    }

    // Scale all color values to [0, 1]
    thrust::device_ptr<vec3> dev_frame_buffer_begin(frame_buffer), dev_frame_buffer_end(frame_buffer + num_pixels);
    float normalizer = thrust::transform_reduce(dev_frame_buffer_begin, dev_frame_buffer_end, max_color(), 0.0f, thrust::maximum<float>());

    int max_color_value = (2 << 15) - 1;  // The max color value for a PPM file
    normalize_color normalize(normalizer, max_color_value);

    std::cout << "P3\n" << nx << " " << ny << "\n" << max_color_value << "\n";
    for (int i = 0; i < num_pixels; ++i)
    {
        std::cout << normalize(frame_buffer[i].r()) << " " << normalize(frame_buffer[i].g()) << " " << normalize(frame_buffer[i].b()) << "\n";
    }

    hipFree(spheres);
    hipFree(sc);
    hipFree(data);
    hipFree(frame_buffer);

    return 0;
}
